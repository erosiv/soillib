#include "hip/hip_runtime.h"
#define HAS_CUDA

#include <soillib/node/flow.hpp>
//#include <soillib/core/texture.hpp>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

#include <iostream>
#include <glm/glm.hpp>

namespace {

__device__ const glm::ivec2 coords[8] = {
  glm::ivec2{-1, 0},
  glm::ivec2{-1, 1},
  glm::ivec2{ 0, 1},
  glm::ivec2{ 1, 1},
  glm::ivec2{ 1, 0},
  glm::ivec2{ 1,-1},
  glm::ivec2{ 0,-1},
  glm::ivec2{-1,-1},
};

__device__ const double dist[8] = {
  1.0,
  HIP_SQRT_TWO,
  1.0,
  HIP_SQRT_TWO,
  1.0,
  HIP_SQRT_TWO,
  1.0,
  HIP_SQRT_TWO
};

__device__ const int dirmap[8] = {
  7, 8, 1, 2, 3, 4, 5, 6,
};

int block(const int elem, const int thread){
  return (elem + thread - 1)/thread;
}

}

//
// Flow Kernel Implementation
//

template<typename T>
__global__ void _flow(soil::buffer_t<T> in, soil::buffer_t<int> out, soil::flat_t<2> index){

  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= in.elem()) return;

  const glm::ivec2 pos = index.unflatten(i);
    
  T diffmax = 0.0f;
  T hvalue = in[i];
  int value = -2;   // default also for nan
  bool pit = true;
  bool has_flow = false;

  for(size_t k = 0; k < 8; ++k){

    const glm::ivec2 coord = coords[k];
    const glm::ivec2 npos = pos + coord;

    if(!index.oob(npos)){
      
      const size_t n = index.flatten(npos);
      const T nvalue = in[n];
      const T ndiff = (hvalue - nvalue)/T(dist[k]);
      
      if(ndiff > diffmax){
        value = k;
        diffmax = ndiff;
      }

      has_flow |= (ndiff > 0.0);
      pit &= (ndiff < 0.0);

      // note: equivalent
      // if(ndiff > 0.0) has_flow = true;
      // if(ndiff >= 0.0) pit = false;

    }

  }

  if(pit) value = -2;
  if(!has_flow && !pit) value = -1;

  if(value >= 0)
    out[i] = dirmap[value];
  else out[i] = value;

}

soil::buffer soil::flow(const soil::buffer& buffer, const soil::index& index) {

  return soil::select(index.type(), [&]<std::same_as<soil::flat_t<2>> I>() {
    return soil::select(buffer.type(), [&]<std::floating_point T>(){

      auto index_t = index.as<I>();
      auto buffer_t = buffer.as<T>();
      buffer_t.to_gpu();

      const int elem = index_t.elem();
      auto out = soil::buffer_t<int>{index_t.elem(), soil::GPU};

      _flow<<<block(elem, 256), 256>>>(buffer_t, out, index_t);
      hipDeviceSynchronize();

      return std::move(soil::buffer(std::move(out)));

    });

  });

}

//
// Direction Kernel Implementation
//

__global__ void _direction(soil::buffer_t<int> in, soil::buffer_t<glm::ivec2> out){

  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= in.elem()) return;

  glm::ivec2 val(0, 0);
  for(size_t k = 0; k < 8; ++k){
    if(in[index] == dirmap[k]){
      val = coords[k];
      break;
    }
  }

  out[index] = val;

}

soil::buffer soil::direction(const soil::buffer& buffer, const soil::index& index){

  return soil::select(index.type(), [&]<std::same_as<soil::flat_t<2>> I>() {
    return soil::select(buffer.type(), [&]<std::same_as<int> T>(){

      auto index_t = index.as<I>();
      auto buffer_t = buffer.as<T>();
      buffer_t.to_gpu();

      const int elem = index_t.elem();
      auto out = soil::buffer_t<soil::ivec2>{index_t.elem(), soil::GPU};

      _direction<<<block(elem, 256), 256>>>(buffer_t, out);
      hipDeviceSynchronize();

      return std::move(soil::buffer(std::move(out)));

    });

  });

}

//
// Accumulation Kernel Implementation
//

template<typename T>
__global__ void _fill(soil::buffer_t<T> buf, const T val){
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= buf.elem()) return;
  buf[index] = val;
}

__global__ void init_randstate(hiprandState* states, const size_t N, const size_t seed) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= N) return;
  hiprand_init(seed, index, 0, &states[index]);
}

__global__ void _accumulate(soil::buffer_t<glm::ivec2> in, soil::buffer_t<int> out, soil::flat_t<2> index, hiprandState* randStates, const int steps, const int N){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= N) return;

  hiprandState* state = &randStates[n];
  glm::ivec2 pos {
    hiprand_uniform(state)*index[0],
    hiprand_uniform(state)*index[1]
  };
  size_t ind = index.flatten(pos);

  for(int s = 0; s < steps; ++s){

    const glm::ivec2 dir = in[ind];
    pos += dir;
    if(dir[0] == 0 && dir[1] == 0)
      break;

    if(index.oob(pos))
      break;

    ind = index.flatten(pos);
    atomicAdd(&(out[ind]), 1);
  }

}

__global__ void _normalize(soil::buffer_t<int> in, soil::buffer_t<double> out, double P){
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= in.elem()) return;
  out[n] = 1.0 + P * (double)in[n];
}

soil::buffer soil::accumulation(const soil::buffer& buffer, const soil::index& index, int iterations, int samples, int steps){

  return soil::select(index.type(), [&]<std::same_as<soil::flat_t<2>> I>() {
    return soil::select(buffer.type(), [&]<std::same_as<soil::ivec2> T>(){

      auto index_t = index.as<I>();
      auto buffer_t = buffer.as<soil::ivec2>();
      buffer_t.to_gpu();

      const size_t elem = index.elem();
      auto out = soil::buffer_t<int>{elem, soil::GPU};
      auto out2 = soil::buffer_t<double>{elem, soil::GPU};

      _fill<<<block(elem, 256), 256>>>(out, 0);

      hiprandState* randStates;
      hipMalloc((void**)&randStates, samples * sizeof(hiprandState));
      init_randstate<<<block(samples, 256), 256>>>(randStates, samples, 0);

      for(int n = 0; n < iterations; ++n)
        _accumulate<<<block(samples, 1024), 1024>>>(buffer_t, out, index_t, randStates, steps, samples);
      hipFree(randStates);

      const double P = double(elem)/double(iterations*samples);
      _normalize<<<block(elem, 256), 256>>>(out, out2, P);
      hipDeviceSynchronize();

      return std::move(soil::buffer(std::move(out2)));

    });

  });

}

//
// Upstream Mask Kernel Implementation
//

__global__ void _upstream(soil::buffer_t<glm::ivec2> in, soil::buffer_t<int> out, glm::ivec2 target, soil::flat_t<2> index, const size_t N){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= N) return;

  bool found = false;
  size_t ind = n;
  glm::ivec2 pos =  index.unflatten(n);
  size_t target_ind = index.flatten(target);

  // note: upper bound is absolute worst-case scenario
  for(int step = 0; step < N; ++step){

    if(ind == target_ind){
      found = true;
      break;
    }

    const glm::ivec2 dir = in[ind];
    pos += dir;
    if(dir[0] == 0 && dir[1] == 0)
      break;

    if(index.oob(pos))
      break;

    ind = index.flatten(pos);

  }

  if(found)
    out[n] = 1;
  else out[n] = 0;

}

// Note: This can potentially be made faster, by batching the upstream kernel execution
// and testing against positions tested in the previous batch (using the output buffer)
// This could be done using a shuffled index buffer (e.g. perfect hash), or using some
// other regular permuation to improve performance. This is not guaranteed to be better.
soil::buffer soil::upstream(const soil::buffer& buffer, const soil::index& index, const glm::ivec2 target){

  return soil::select(index.type(), [&]<std::same_as<soil::flat_t<2>> I>() {
    return soil::select(buffer.type(), [&]<std::same_as<soil::ivec2> T>(){

      auto index_t = index.as<I>();
      auto buffer_t = buffer.as<soil::ivec2>();
      buffer_t.to_gpu();

      const size_t elem = index_t.elem();
      auto out = soil::buffer_t<int>{elem, soil::GPU};

      _fill<<<block(elem, 256), 256>>>(out, -1);
      if(!index_t.oob(target)){
        _upstream<<<block(elem, 256), 256>>>(buffer_t, out, target, index_t, elem);
      }
      hipDeviceSynchronize();

      return std::move(soil::buffer(std::move(out)));

    });

  });

}

//
// Upstream Distance Kernel Implementation
//

__global__ void _distance(soil::buffer_t<glm::ivec2> in, soil::buffer_t<int> out, glm::ivec2 target, soil::flat_t<2> index, const size_t N){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= N) return;

  size_t ind = n;
  glm::ivec2 pos =  index.unflatten(n);
  size_t target_ind = index.flatten(target);

  // note: upper bound is absolute worst-case scenario
  for(int step = 0; step < N; ++step){

    if(ind == target_ind){
      out[n] = step;
      break;
    }

    const glm::ivec2 dir = in[ind];
    pos += dir;
    if(dir[0] == 0 && dir[1] == 0)
      break;

    if(index.oob(pos))
      break;

    ind = index.flatten(pos);

  }

}

soil::buffer soil::distance(const soil::buffer& buffer, const soil::index& index, const glm::ivec2 target){

  return soil::select(index.type(), [&]<std::same_as<soil::flat_t<2>> I>() {
    return soil::select(buffer.type(), [&]<std::same_as<soil::ivec2> T>(){

      auto index_t = index.as<I>();
      auto buffer_t = buffer.as<T>();
      buffer_t.to_gpu();

      const size_t elem = index.elem();
      auto out = soil::buffer_t<int>{elem, soil::GPU};

      _fill<<<block(elem, 256), 256>>>(out, 2); // unknown state...
      if(!index_t.oob(target)){
        _distance<<<block(elem, 256), 256>>>(buffer_t, out, target, index_t, elem);
      }
      hipDeviceSynchronize();

      return std::move(soil::buffer(std::move(out)));

    });
  });

}

// note: move this to a different file