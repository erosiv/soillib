#include "hip/hip_runtime.h"
#define HAS_CUDA

#include <soillib/node/flow.hpp>
#include <soillib/core/texture.hpp>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

#include <iostream>
#include <glm/glm.hpp>

/*
Potential Concept for Improving Performance:
Instead of letting each possible path run in a batch
until completion, we should run then for a fixed number
of steps and then check which paths have not yet terminated.

Those that have not yet terminated are re-entered into a queue
and we re-execute for only those guys... That should basically
give an overall performance boost.
*/

namespace {

__device__ const glm::ivec2 coords[8] = {
  glm::ivec2{-1, 0},
  glm::ivec2{-1, 1},
  glm::ivec2{ 0, 1},
  glm::ivec2{ 1, 1},
  glm::ivec2{ 1, 0},
  glm::ivec2{ 1,-1},
  glm::ivec2{ 0,-1},
  glm::ivec2{-1,-1},
};

__device__ const double dist[8] = {
  1.0,
  HIP_SQRT_TWO,
  1.0,
  HIP_SQRT_TWO,
  1.0,
  HIP_SQRT_TWO,
  1.0,
  HIP_SQRT_TWO
};

__device__ const int dirmap[8] = {
  7, 8, 1, 2, 3, 4, 5, 6,
};

int block(const int elem, const int thread){
  return (elem + thread - 1)/thread;
}

}

//
// Flow Kernel Implementation
//

template<typename T>
__global__ void _flow(soil::buffer_t<T> in, soil::buffer_t<int> out, soil::flat_t<2> index){

  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= in.elem()) return;

  const glm::ivec2 pos = index.unflatten(i);
    
  T diffmax = 0.0f;
  T hvalue = in[i];
  int value = -2;   // default also for nan
  bool pit = true;
  bool has_flow = false;

  for(size_t k = 0; k < 8; ++k){

    const glm::ivec2 coord = coords[k];
    const glm::ivec2 npos = pos + coord;

    if(!index.oob(npos)){
      
      const size_t n = index.flatten(npos);
      const T nvalue = in[n];
      const T ndiff = (hvalue - nvalue)/T(dist[k]);
      
      if(ndiff > diffmax){
        value = k;
        diffmax = ndiff;
      }

      has_flow |= (ndiff > 0.0);
      pit &= (ndiff < 0.0);

      // note: equivalent
      // if(ndiff > 0.0) has_flow = true;
      // if(ndiff >= 0.0) pit = false;

    }

  }

  if(pit) value = -2;
  if(!has_flow && !pit) value = -1;

  if(value >= 0)
    out[i] = dirmap[value];
  else out[i] = value;

}

soil::buffer soil::flow(const soil::buffer& buffer, const soil::index& index) {

  return soil::select(index.type(), [&]<std::same_as<soil::flat_t<2>> I>() {
    return soil::select(buffer.type(), [&]<std::floating_point T>(){

      auto index_t = index.as<I>();
      auto buffer_t = buffer.as<T>();
      buffer_t.to_gpu();

      const int elem = index_t.elem();
      auto out = soil::buffer_t<int>{index_t.elem(), soil::GPU};

      _flow<<<block(elem, 256), 256>>>(buffer_t, out, index_t);
      hipDeviceSynchronize();

      return std::move(soil::buffer(std::move(out)));

    });

  });

}

//
// Direction Kernel Implementation
//

__device__ soil::ivec2 _get_dir(const int flow){
  for(size_t k = 0; k < 8; ++k){
    if(flow == dirmap[k])
      return coords[k];
  }
  return {0, 0};
}

__global__ void _direction(soil::buffer_t<int> in, soil::buffer_t<glm::ivec2> out){
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= in.elem()) return;
  out[index] = _get_dir(in[index]);
}

soil::buffer soil::direction(const soil::buffer& buffer, const soil::index& index){

  return soil::select(index.type(), [&]<std::same_as<soil::flat_t<2>> I>() {
    return soil::select(buffer.type(), [&]<std::same_as<int> T>(){

      auto index_t = index.as<I>();
      auto buffer_t = buffer.as<T>();
      buffer_t.to_gpu();

      const int elem = index_t.elem();
      auto out = soil::buffer_t<soil::ivec2>{index_t.elem(), soil::GPU};

      _direction<<<block(elem, 256), 256>>>(buffer_t, out);
      hipDeviceSynchronize();

      return std::move(soil::buffer(std::move(out)));

    });

  });

}

//
// Accumulation Kernel Implementation
//

template<typename T>
__global__ void _fill(soil::buffer_t<T> buf, const T val){
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= buf.elem()) return;
  buf[index] = val;
}

__global__ void init_randstate(hiprandState* states, const size_t N, const size_t seed) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= N) return;
  hiprand_init(seed, index, 0, &states[index]);
}

__global__ void _accumulate(soil::texture<int> texture, soil::buffer_t<int> out, soil::flat_t<2> index, hiprandState* randStates, const int steps, const int N){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= N) return;

  hiprandState* state = &randStates[n];
  size_t ind = hiprand_uniform(state)*index.elem();
  glm::ivec2 pos = index.unflatten(ind);

  for(int s = 0; s < steps; ++s){

    const int flow = texture[soil::vec2(pos)];
    const glm::ivec2 dir = _get_dir(flow);
    pos += dir;
    if(dir[0] == 0 && dir[1] == 0)
      break;

    if(index.oob(pos))
      break;

    ind = index.flatten(pos);
    atomicAdd(&(out[ind]), 1);
  }

}

__global__ void _normalize(soil::buffer_t<int> in, soil::buffer_t<double> out, double P){
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= in.elem()) return;
  out[n] = 1.0 + P * (double)in[n];
}

soil::buffer soil::accumulation(const soil::buffer& buffer, const soil::index& index, int iterations, int samples, int steps){

  return soil::select(index.type(), [&]<std::same_as<soil::flat_t<2>> I>() {
    return soil::select(buffer.type(), [&]<std::same_as<int> T>(){

      auto index_t = index.as<I>();
      auto buffer_t = buffer.as<T>();
      buffer_t.to_gpu();

      texture<int> texture(buffer_t, index_t);

      const size_t elem = index.elem();
      auto out = soil::buffer_t<int>{elem, soil::GPU};
      auto out2 = soil::buffer_t<double>{elem, soil::GPU};

      _fill<<<block(elem, 256), 256>>>(out, 0);

      hiprandState* randStates;
      hipMalloc((void**)&randStates, samples * sizeof(hiprandState));
      init_randstate<<<block(samples, 256), 256>>>(randStates, samples, 0);

      for(int n = 0; n < iterations; ++n)
        _accumulate<<<block(samples, 512), 512>>>(texture, out, index_t, randStates, steps, samples);

      const double P = double(elem)/double(iterations*samples);
      _normalize<<<block(elem, 256), 256>>>(out, out2, P);

      hipFree(randStates);
      hipDeviceSynchronize();

      return std::move(soil::buffer(std::move(out2)));

    });

  });

}

//
// Upstream Mask Kernel Implementation
//



namespace {

//! \todo make this robust for non-regular tile shapes (test)
__device__ soil::ivec2 tile_unflatten(const unsigned int ind, const int h){

  constexpr int tile_w = 8;
  constexpr int tile_h = 8;
  constexpr int tile_s = tile_w * tile_h;

  // Binned Tile Index, Tile Position

  unsigned int tile_ind = ind / tile_s;
  unsigned int tile_x = tile_w * (tile_ind / (h / tile_h));
  unsigned int tile_y = tile_h * (tile_ind % (h / tile_h));

  unsigned int tile_pos = ind % tile_s;
  unsigned int x = tile_x + tile_pos / tile_h;
  unsigned int y = tile_y + tile_pos % tile_h;

  return soil::ivec2(x, y);

}

}

__global__ void _upstream(const soil::buffer_t<glm::ivec2> in, soil::buffer_t<int> out, glm::ivec2 target, soil::flat_t<2> index, const size_t N){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= N) return;

  soil::ivec2 pos = tile_unflatten(n, index[0]);
  size_t ind = index.flatten(pos);
  const size_t ind0 = ind;
  soil::ivec2 dir;

  int found = 0;

  size_t target_ind = index.flatten(target);

  // note: upper bound is absolute worst-case scenario
  while(ind != target_ind && !index.oob(pos)){

    dir = in[ind];
    if(dir[0] == 0 && dir[1] == 0)
      break;

    pos += dir;
    ind = index.flatten(pos);
    if(ind == target_ind){
      found = 1;
    }

  }

  out[ind0] |= found;
}

// Note: This can potentially be made faster, by batching the upstream kernel execution
// and testing against positions tested in the previous batch (using the output buffer)
// This could be done using a shuffled index buffer (e.g. perfect hash), or using some
// other regular permuation to improve performance. This is not guaranteed to be better.
soil::buffer soil::upstream(const soil::buffer& buffer, const soil::index& index, const glm::ivec2 target){

  return soil::select(index.type(), [&]<std::same_as<soil::flat_t<2>> I>() {
    return soil::select(buffer.type(), [&]<std::same_as<soil::ivec2> T>(){

      auto index_t = index.as<I>();
      auto buffer_t = buffer.as<soil::ivec2>();
      buffer_t.to_gpu();

      const size_t elem = index_t.elem();
      auto out = soil::buffer_t<int>{elem, soil::GPU};

      _fill<<<block(elem, 256), 256>>>(out, 0);
      if(!index_t.oob(target)){
        _upstream<<<block(elem, 512), 512>>>(buffer_t, out, target, index_t, elem);
      }
      hipDeviceSynchronize();

      return std::move(soil::buffer(std::move(out)));

    });

  });

}

//
// Upstream Distance Kernel Implementation
//

__global__ void _distance(soil::buffer_t<glm::ivec2> in, soil::buffer_t<int> out, glm::ivec2 target, soil::flat_t<2> index, const size_t N){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= N) return;

  soil::ivec2 pos = tile_unflatten(n, index[0]);
  size_t ind = index.flatten(pos);
  const size_t ind0 = ind;
  soil::ivec2 dir;

  size_t target_ind = index.flatten(target);

  // note: upper bound is absolute worst-case scenario
  for(int step = 0; step < N; ++step){

    if(ind == target_ind){
      out[ind0] = step;
      break;
    }

    const glm::ivec2 dir = in[ind];
    pos += dir;
    if(dir[0] == 0 && dir[1] == 0)
      break;

    if(index.oob(pos))
      break;

    ind = index.flatten(pos);

  }

}

soil::buffer soil::distance(const soil::buffer& buffer, const soil::index& index, const glm::ivec2 target){

  return soil::select(index.type(), [&]<std::same_as<soil::flat_t<2>> I>() {
    return soil::select(buffer.type(), [&]<std::same_as<soil::ivec2> T>(){

      auto index_t = index.as<I>();
      auto buffer_t = buffer.as<T>();
      buffer_t.to_gpu();

      const size_t elem = index.elem();
      auto out = soil::buffer_t<int>{elem, soil::GPU};

      _fill<<<block(elem, 256), 256>>>(out, -1); // unknown state...
      if(!index_t.oob(target)){
        _distance<<<block(elem, 512), 512>>>(buffer_t, out, target, index_t, elem);
      }
      hipDeviceSynchronize();

      return std::move(soil::buffer(std::move(out)));

    });
  });

}

// note: move this to a different file