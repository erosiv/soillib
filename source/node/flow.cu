#include "hip/hip_runtime.h"
#define HAS_CUDA

#include <soillib/node/flow.hpp>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

#include <iostream>
#include <glm/glm.hpp>

namespace {

__device__ const glm::ivec2 coords[8] = {
  glm::ivec2{-1, 0},
  glm::ivec2{-1, 1},
  glm::ivec2{ 0, 1},
  glm::ivec2{ 1, 1},
  glm::ivec2{ 1, 0},
  glm::ivec2{ 1,-1},
  glm::ivec2{ 0,-1},
  glm::ivec2{-1,-1},
};

__device__ const double dist[8] = {
  1.0,
  HIP_SQRT_TWO,
  1.0,
  HIP_SQRT_TWO,
  1.0,
  HIP_SQRT_TWO,
  1.0,
  HIP_SQRT_TWO
};

__device__ const int dirmap[8] = {
  7, 8, 1, 2, 3, 4, 5, 6,
};

}

__global__ void _flow(soil::buffer_t<double> in, soil::buffer_t<int> out, soil::flat_t<2> index){

  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= in.elem()) return;

  const glm::ivec2 pos = index.unflatten(i);
    
  double diffmax = 0.0f;
  double hvalue = in[i];
  int value = -2;   // default also for nan
  bool pit = true;
  bool has_flow = false;

  for(size_t k = 0; k < 8; ++k){

    const glm::ivec2 coord = coords[k];
    const glm::ivec2 npos = pos + coord;

    if(!index.oob(npos)){
      
      const size_t n = index.flatten(npos);
      const double nvalue = in[n];
      const double ndiff = (hvalue - nvalue)/dist[k];
      
      if(ndiff > diffmax){
        value = k;
        diffmax = ndiff;
      }

      has_flow |= (ndiff > 0.0);
      pit &= (ndiff < 0.0);

      // note: equivalent
      // if(ndiff > 0.0) has_flow = true;
      // if(ndiff >= 0.0) pit = false;

    }

  }

  if(pit) value = -2;
  if(!has_flow && !pit) value = -1;

  if(value >= 0)
    out[i] = dirmap[value];
  else out[i] = value;

}

__global__ void _direction(soil::buffer_t<int> in, soil::buffer_t<glm::ivec2> out){

  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= in.elem()) return;

  glm::ivec2 val(0, 0);
  for(size_t k = 0; k < 8; ++k){
    if(in[index] == dirmap[k]){
      val = coords[k];
      break;
    }
  }

  out[index] = val;

}

template<typename T>
__global__ void _fill(soil::buffer_t<T> buf, const T val){
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < buf.size())
    buf[index] = val;
}

__global__ void init_randstate(hiprandState* states, const size_t N, const size_t seed) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= N) return;
  hiprand_init(seed, index, 0, &states[index]);
}

__global__ void _accumulate(soil::buffer_t<glm::ivec2> in, soil::buffer_t<int> out, soil::flat_t<2> index, hiprandState* randStates, const int steps, const int N){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= N) return;

  hiprandState* state = &randStates[n];
  glm::ivec2 pos {
    hiprand_uniform(state)*index[0],
    hiprand_uniform(state)*index[1]
  };
  size_t ind = index.flatten(pos);

  for(size_t s = 0; s < steps; ++s){

    const glm::ivec2 dir = in[ind];
    pos += dir;
    if(dir[0] == 0 && dir[1] == 0)
      break;

    if(index.oob(pos))
      break;

    ind = index.flatten(pos);
    atomicAdd(&(out[ind]), 1);
  }

}

__global__ void _normalize(soil::buffer_t<int> in, soil::buffer_t<double> out, double P){
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= in.elem()) return;
  out[n] = 1.0 + P * (double)in[n];
}

soil::buffer soil::flow::full() const {

  const int elem = index.elem();
  auto in = this->buffer.as<double>();
  in.to_gpu();

  const int thread = 1024;
  const int block = (elem + thread - 1)/thread;
  
  auto out = buffer_t<int>{index.elem(), GPU};
  _flow<<<block, thread>>>(in, out, index);

  return std::move(soil::buffer(std::move(out)));

}

soil::buffer soil::direction::full() const {

  const int elem = index.elem();
  auto in = this->buffer.as<int>();
  in.to_gpu();

  const int thread = 1024;
  const int block = (elem + thread - 1)/thread;

  auto out = buffer_t<ivec2>{index.elem(), GPU};
  _direction<<<block, thread>>>(in, out);

  return std::move(soil::buffer(std::move(out)));

}

__global__ void _upstream(soil::buffer_t<glm::ivec2> in, soil::buffer_t<int> out, glm::ivec2 target, soil::flat_t<2> index, const size_t N){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= N) return;

  bool found = false;
  size_t ind = n;
  glm::ivec2 pos =  index.unflatten(n);
  size_t target_ind = index.flatten(target);

  // note: upper bound is absolute worst-case scenario
  for(int step = 0; step < N; ++step){

    if(ind == target_ind){
      found = true;
      break;
    }

    const glm::ivec2 dir = in[ind];
    pos += dir;
    if(dir[0] == 0 && dir[1] == 0)
      break;

    if(index.oob(pos))
      break;

    ind = index.flatten(pos);

  }

  if(found)
    out[n] = 1;
  else out[n] = 0;

}

soil::buffer soil::upstream::full() const {

  // I suppose the ideal solution is to do a random order...
  // how do we generate a list of random order?
  // we could do a perfect hash instead...
  // or we could really just use a random number generator
  // and speculate that we get sufficient hits.
  // this determinism is not necessarily suited to GPU computation...
  // unless I can perform some kind of radix sort on the elements
  // but that would require some kind of tree index which I might not have.
  // the sort would basically be: am I above or below a value...
  // but we would sort over the set of indices...

  /*
  so if we did a sort that would effectively yield the solution...
  for now, we will just do it in order and see if shuffling improves performance at all later...
  */

  // Input Direction Buffer!
  const size_t elem = index.elem();
  auto in = this->buffer.as<ivec2>();
  in.to_gpu();

  auto out = buffer_t<int>{elem, GPU};
  int thread = 1024;
  int block = (elem + thread - 1)/thread;
  _fill<<<block, thread>>>(out, 2); // unknown state...
  
  thread = 1024;
  block = (elem + thread - 1)/thread;
  _upstream<<<block, thread>>>(in, out, target, index, elem);

  return std::move(soil::buffer(std::move(out)));

}

soil::buffer soil::accumulation::full() const {

  const size_t elem = index.elem();
  auto in = this->buffer.as<ivec2>();
  in.to_gpu();

  auto out = buffer_t<int>{elem, GPU};
  int thread = 1024;
  int block = (elem + thread - 1)/thread;
  _fill<<<block, thread>>>(out, 0);

  auto out2 = buffer_t<double>{elem, GPU};
  
  hiprandState* randStates;
  hipMalloc((void**)&randStates, this->samples * sizeof(hiprandState));

  thread = 1024;
  block = (this->samples + thread - 1)/thread;

  init_randstate<<<block, thread>>>(randStates, this->samples, 0);

  for(int n = 0; n < this->iterations; ++n)
    _accumulate<<<block, thread>>>(in, out, index, randStates, this->steps, this->samples);
  hipFree(randStates);

  thread = 1024;
  block = (elem + thread - 1)/thread;
  const double P = double(elem)/double(iterations*samples);
  _normalize<<<block, thread>>>(out, out2, P);

  return std::move(soil::buffer(std::move(out2)));

}

// note: move this to a different file
