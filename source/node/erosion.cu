#include "hip/hip_runtime.h"
#ifndef SOILLIB_NODE_EROSION_CU
#define SOILLIB_NODE_EROSION_CU
#define HAS_CUDA

#include <soillib/node/erosion.hpp>
#include <soillib/util/error.hpp>
#include <soillib/node/lerp.cu>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <iostream>

#include "erosion_thermal.cu"

namespace soil {

namespace {

template<typename T>
struct sample_t {
  glm::ivec2 pos;
  T value;
  bool oob = true;
};

template<typename T, typename I>
__device__ void gather(const soil::buffer_t<T> &buffer_t, const I index, glm::ivec2 p, sample_t<T> px[5], sample_t<T> py[5]) {
  for (int i = 0; i < 5; ++i) {

    const glm::ivec2 pos_x = p + glm::ivec2(-2 + i, 0);
    if (!index.oob(pos_x)) {
      px[i].oob = false;
      px[i].pos = pos_x;

      const size_t ind = index.flatten(pos_x);
      px[i].value = buffer_t[ind];
    }

    const glm::ivec2 pos_y = p + glm::ivec2(0, -2 + i);
    if (!index.oob(pos_y)) {
      py[i].oob = false;
      py[i].pos = pos_y;

      const size_t ind = index.flatten(pos_y);
      py[i].value = buffer_t[ind];
    }
  }
}

template<std::floating_point T>
__device__ glm::vec2 gradient_detailed(sample_t<T> px[5], sample_t<T> py[5]) {

  glm::vec2 g = glm::vec2(0, 0);

  // X-Element
  if (!px[0].oob && !px[4].oob)
    g.x = (1.0f * px[0].value - 8.0f * px[1].value + 8.0f * px[3].value - 1.0f * px[4].value) / 12.0f;

  else if (!px[0].oob && !px[3].oob)
    g.x = (1.0f * px[0].value - 6.0f * px[1].value + 3.0f * px[2].value + 2.0f * px[3].value) / 6.0f;

  else if (!px[0].oob && !px[2].oob)
    g.x = (1.0f * px[0].value - 4.0f * px[1].value + 3.0f * px[2].value) / 2.0f;

  else if (!px[1].oob && !px[4].oob)
    g.x = (-2.0f * px[1].value - 3.0f * px[2].value + 6.0f * px[3].value - 1.0f * px[4].value) / 6.0f;

  else if (!px[2].oob && !px[4].oob)
    g.x = (-3.0f * px[2].value + 4.0f * px[3].value - 1.0f * px[4].value) / 2.0f;

  else if (!px[1].oob && !px[3].oob)
    g.x = (-1.0f * px[1].value + 1.0f * px[3].value) / 2.0f;

  else if (!px[2].oob && !px[3].oob)
    g.x = (-1.0f * px[2].value + 1.0f * px[3].value) / 1.0f;

  else if (!px[1].oob && !px[2].oob)
    g.x = (-1.0f * px[1].value + 1.0f * px[2].value) / 1.0f;

  // Y-Element

  if (!py[0].oob && !py[4].oob)
    g.y = (1.0f * py[0].value - 8.0f * py[1].value + 8.0f * py[3].value - 1.0f * py[4].value) / 12.0f;

  else if (!py[0].oob && !py[3].oob)
    g.y = (1.0f * py[0].value - 6.0f * py[1].value + 3.0f * py[2].value + 2.0f * py[3].value) / 6.0f;

  else if (!py[0].oob && !py[2].oob)
    g.y = (1.0f * py[0].value - 4.0f * py[1].value + 3.0f * py[2].value) / 2.0f;

  else if (!py[1].oob && !py[4].oob)
    g.y = (-2.0f * py[1].value - 3.0f * py[2].value + 6.0f * py[3].value - 1.0f * py[4].value) / 6.0f;

  else if (!py[2].oob && !py[4].oob)
    g.y = (-3.0f * py[2].value + 4.0f * py[3].value - 1.0f * py[4].value) / 2.0f;

  else if (!py[1].oob && !py[3].oob)
    g.y = (-1.0f * py[1].value + 1.0f * py[3].value) / 2.0f;

  else if (!py[2].oob && !py[3].oob)
    g.y = (-1.0f * py[2].value + 1.0f * py[3].value) / 1.0f;

  else if (!py[1].oob && !py[2].oob)
    g.y = (-1.0f * py[1].value + 1.0f * py[2].value) / 1.0f;

  return g;
}

__device__ vec2 gradient(const model_t& model, const vec2 pos){

  sample_t<float> px[5], py[5];
  gather<float, soil::flat_t<2>>(model.height, model.index, ivec2(pos), px, py);
  return gradient_detailed<float>(px, py);

}

__device__ float sigmoid(float x) {
  return x / sqrt(1.0f + x*x);
}

int block(const int elem, const int thread){
  return (elem + thread - 1)/thread;
}

}

//
// Randstate and Estimate Initialization / Filtering
//

__global__ void init_randstate(hiprandState* states, const size_t N, const size_t seed, const size_t offset) {

  const unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= N) return;
  
  hiprand_init(seed, n, 2*offset, &states[n]); // scale by 2 because we take two random samples per iteration

}

__global__ void reset(model_t model){
  
  const unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= model.elem) return;
  
  // Reset Estimation Buffers

  model.discharge_track[n] = 0.0f;
  model.suspended_track[n] = 0.0f;
  model.momentum_track[n] = vec2(0.0f);
  model.equilibrium_track[n] = 0.0f;

}

template<typename T>
__device__ T mix(T a, T b, float w){
  return (1.0f-w)*a + w*b;
}

__global__ void filter(model_t model, const param_t param){

  const unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= model.elem) return;

  // Apply Simple Exponential Filter to Noisy Estimates

  model.discharge[n] = mix<float>(model.discharge[n], model.discharge_track[n], param.lrate);
  model.momentum[n] = mix<vec2>(model.momentum[n], model.momentum_track[n], param.lrate);

  model.suspended[n] = mix<float>(model.suspended[n], model.suspended_track[n], 0.9f);
  model.equilibrium[n] = mix<float>(model.equilibrium[n], model.equilibrium_track[n], 0.9f);

}

//
// Erosion Kernels
//

__device__ float equ_frac(const model_t& model, vec2 pos, vec2 npos, const param_t param){

  const int find = model.index.flatten(pos);
  const int nind = model.index.flatten(npos);

  float h0 = model.height[find];
  float h1 = h0 - param.exitSlope; 
  if(!model.index.oob(npos)){
    h1 = model.height[nind];
  }

  const float discharge = glm::max(0.0f, model.discharge[find]);  // Discharge Volume
  const float slope = (h0 - h1);                  // Local Slope

  return glm::max(slope, 0.0f) * param.entrainment * log(1.0f + discharge);

}

__global__ void solve(model_t model, hiprandState* randStates, const size_t N, const param_t param){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= N) return;

  //
  // Parameters
  // Note: Scale-Normalize Values
  //

  const float mu = param.momentumTransfer;
  const float g = param.gravity;
  const float k = param.depositionRate;

  //
  // Initial Condition
  //
  
  // Trajectory and Integration State

  const float P = float(model.elem)/float(N); // Sample Probability
  hiprandState* randState = &randStates[ind];
  vec2 pos = vec2{
    hiprand_uniform(randState)*float(model.index[0]),
    hiprand_uniform(randState)*float(model.index[1])
  };

  int find = model.index.flatten(pos);

  float vol = 1.0f;
  float sed = 0.0f;

  const vec2 grad = gradient(model, pos);
  const vec3 normal = glm::normalize(vec3(-grad.x, -grad.y, 1.0f));
  const vec2 average_speed = (model.momentum[find]) / (1.0f + model.discharge[find]);
  vec2 speed = g * vec2(normal.x, normal.y) + (mu / vol) * average_speed;

  vec2 dspeed = speed;

  // Solution Loop:
  //  Solve Conservation Law along Characteristic
  //  Generated by the Flow.

  for(size_t age = 0; age < param.maxage; ++age){

    // Termination Conditions

    if(model.index.oob(pos))      return;
    if(vol < param.minVol)        return;
    if(glm::length(speed) < 1E-4) return;

    //
    // Execute Integration
    //

    // Flow Integration / Trajectory

    vec2 nspeed = speed;
    vec2 npos = pos;

    // Viscosity Contribution

    const vec2 average_speed = (model.momentum[find] + vol * speed) / (1.0f + model.discharge[find] + vol);
    nspeed += mu * (average_speed - speed);

    // Gravity Contribution

    const vec2 grad = gradient(model, pos);
    const vec3 normal = glm::normalize(vec3(-grad.x, -grad.y, 1.0f));
    nspeed += g * vec2(normal.x, normal.y);

    //
    // Time-Step Normalization
    //

    // Note: Here we should see if we can use the length of the speed
    //  vector as the inverse of the time-step. That would help scale
    //  things correctly.
    //  Additionally, we should limit the magnitude of the velocity,
    //  because it does have the change to run-away despite the viscosity.

    if(glm::length(nspeed) > 0.0){
      npos += sqrt(2.0f)*glm::normalize(nspeed);
    } else {
      // note: if the position becomes the same,
      // slope will also be zero
      // meaning equilibrium drops to zero
      // which could cause a chain reaction of deposition
      break;
    }

    //
    // Mass-Transfer
    //

    const float equilibrium = vol * equ_frac(model, pos, npos, param);

    //
    // Accumulate Estimated Values
    //

    // Note: Accumulation Occurds at Current Position

    atomicAdd(&model.discharge_track[find], P*vol);
    atomicAdd(&model.momentum_track[find].x, P*vol*dspeed.x);
    atomicAdd(&model.momentum_track[find].y, P*vol*dspeed.y);

    // Note: Both of these work but are slightly different. Find out why!
    
    //atomicAdd(&model.equilibrium_track[find], equilibrium);
    //atomicAdd(&model.suspended_track[find], sed);
    atomicAdd(&model.height[find], -k*(equilibrium - sed));

    //
    // Integrate Sub-Solution Quantities
    //

    vol *= (1.0f - param.evapRate);
    dspeed += - (mu / vol)*dspeed;
    sed += k * (equilibrium - sed);

    // Update Position at next Position?
    // We do this because technically,
    // we have moved forward to where
    // the velocity has changed as specified.

    //
    // Update Trajectory
    //

    pos = npos;
    speed = nspeed;
    find = model.index.flatten(pos);

  }

}

__global__ void apply_height(model_t model, const param_t param){

  const unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= model.elem) return;

  // what is the correct way to scale this difference value?
  // since equilibrium is effectively weighted by vol,
  // and sediment is scaled by volume as well in theory...
  // so one is an average concentration, the other is an
  // an average equilibrium concentration.

  const float k = glm::clamp(param.depositionRate, 0.0f, 1.0f);
  const float equilibrium = model.equilibrium[n];
  const float discharge = model.discharge[n];
  const float sediment = model.suspended[n];
  if(discharge > 0.0f){
    model.height[n] += -param.hscale*k*(equilibrium - sediment)/(discharge);
  }

}

//
// Erosion Function
//

void gpu_erode(model_t& model, const param_t param, const size_t steps, const size_t n_samples){

  if(model.height.host() != soil::host_t::GPU){
    throw soil::error::mismatch_host(soil::host_t::GPU, model.height.host());
  }

  if(model.discharge.host() != soil::host_t::GPU){
    throw soil::error::mismatch_host(soil::host_t::GPU, model.discharge.host());
  }

  if(model.momentum.host() != soil::host_t::GPU){
    throw soil::error::mismatch_host(soil::host_t::GPU, model.momentum.host());
  }
  
  //
  // Initialize Rand-State Buffer (One Per Sample)
  //

  // note: the offset in the sequence should be number of times rand is sampled
  // that way the sampling procedure becomes deterministic
  hiprandState* randStates;
  hipMalloc((void**)&randStates, n_samples * sizeof(hiprandState));
  init_randstate<<<block(n_samples, 512), 512>>>(randStates, n_samples, 0, model.age);
  hipDeviceSynchronize();

  //
  // Estimate Buffers
  //

  // Note: Extract this Allocation
  auto buf_discharge = soil::buffer_t<float>(model.discharge.elem(), soil::host_t::GPU);
  auto buf_suspended = soil::buffer_t<float>(model.discharge.elem(), soil::host_t::GPU);
  auto buf_momentum = soil::buffer_t<vec2>(model.discharge.elem(), soil::host_t::GPU);
  model.discharge_track = buf_discharge;
  model.suspended_track = buf_suspended;
  model.momentum_track = buf_momentum;

  auto buf_equilibrium = soil::buffer_t<float>(model.discharge.elem(), soil::host_t::GPU);
  auto buf_equilibrium_track = soil::buffer_t<float>(model.discharge.elem(), soil::host_t::GPU);
  model.equilibrium = buf_equilibrium;
  model.equilibrium_track = buf_equilibrium_track;

  //
  // Execute Solution
  //

  for(size_t step = 0; step < steps; ++step){

    //
    // Reset, Solve, Filter, Apply
    //

    reset<<<block(model.elem, 1024), 1024>>>(model);
    hipDeviceSynchronize();

    solve<<<block(n_samples, 512), 512>>>(model, randStates, n_samples, param);
    hipDeviceSynchronize();
 
    filter<<<block(model.elem, 1024), 1024>>>(model, param);
    hipDeviceSynchronize();

    //
    // Apply Height-Map Updates
    //

    // apply the suspension difference...
    // apply_height<<<block(model.elem, 1024), 1024>>>(model, param);

    // atomic add operations might still be coming in -
    // we have to be done before cascading or this fails...
    // we can't be computing the differences before they are determined...

    compute_cascade<<<block(model.elem, 1024), 1024>>>(model, model.discharge_track, param);
    apply_cascade<<<block(model.elem, 1024), 1024>>>(model, model.discharge_track, param);
    hipDeviceSynchronize();

    // Increment Model Age for Rand-State Initialization
    model.age++;

  }

  hipFree(randStates);

}

} // end of namespace soil

#endif