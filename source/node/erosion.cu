#include "hip/hip_runtime.h"
#ifndef SOILLIB_NODE_EROSION_CU
#define SOILLIB_NODE_EROSION_CU
#define HAS_CUDA

#include <soillib/node/erosion.hpp>
#include <soillib/util/error.hpp>
#include <soillib/node/lerp.cu>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <iostream>

namespace soil {

namespace {

int block(const int elem, const int thread){
  return (elem + thread - 1)/thread;
}

}

__global__ void init_randstate(hiprandState* states, const size_t N, const size_t seed) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= N) return;
  hiprand_init(seed, index, 0, &states[index]);
}

__global__ void spawn(buffer_t<vec2> pos_buf, hiprandState* randStates, flat_t<2> index){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= pos_buf.size()) return;

  hiprandState* randState = &randStates[n];
  vec2 pos {
    hiprand_uniform(randState)*float(index[0] - 1),
    hiprand_uniform(randState)*float(index[1] - 1)
  };

  pos_buf[n] = pos;

}

template<typename T>
__global__ void fill(soil::buffer_t<T> buf, const T val){
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < buf.size())
    buf[index] = val;
}

__global__ void descend(const soil::buffer_t<float> height, const soil::flat_t<2> index, soil::buffer_t<vec2> pos, soil::buffer_t<vec2> speed){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= pos.elem()) return;

  if(oob(pos[ind], index)) 
    return;

  const lerp_t<float> lerp = gather(height, index, pos[ind]);
  const vec2 grad = lerp.grad();
  const vec3 n = glm::normalize(vec3(-grad.x, -grad.y, 1.0));

//  vec2 s = speed[ind];
//  s += 2.0f * vec2(n);
//  if(glm::length(s) > 0.0f){
//    s = sqrtf(2.0f) * glm::normalize(s);
//  }
  vec2 s = vec2(n);
//  vec2 s = glm::normalize(vec2(n)); // pure normal descent...
  speed[ind] = s;
  pos[ind] += s;

}

__global__ void _discharge(soil::buffer_t<float> discharge, const soil::flat_t<2> index, soil::buffer_t<vec2> pos, soil::buffer_t<float> vol_b){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= pos.elem()) return;

  if(oob(pos[ind], index)) 
    return;

  const int find = index.flatten(pos[ind]);
  const float vol = vol_b[ind];
  atomicAdd(&discharge[find], vol);

}

__global__ void transfer(soil::buffer_t<float> height, const soil::flat_t<2> index, soil::buffer_t<vec2> pos_b, soil::buffer_t<vec2> speed_b, soil::buffer_t<float> vol_b, soil::buffer_t<float> sed_b){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= pos_b.elem()) return;

  const float evapRate = 0.001f;
  const float depositionRate = 0.05f;

  const vec2 speed = speed_b[ind];  // Current Speed
  const vec2 pos1 = pos_b[ind];     // Current Position
  const vec2 pos0 = pos1 - speed;   // Old Position

//  if(glm::length(speed) < 1e-6){
//    return;
//  }

  float h0, h1;
  if(index.oob(pos0)) return;

  h0 = height[index.flatten(pos0)];

  if(index.oob(pos1)){
    h1 = 0.99f*h0;
  } else {
    h1 = height[index.flatten(pos1)];
  }

//  if(oob(pos0, index)) return;

  // Compute Height Difference

  /*
  float h0, h1;
  lerp_t<float> lerp0 = gather(height, index, pos0);
  h0 = lerp0.val();
  
  if(!oob(pos1, index)){
    lerp_t<float> lerp1 = gather(height, index, pos1);
    h1 = lerp1.val();
  } else {
    h1 = 0.99f * h0;
  }
  */

  // Compute Equilibrium Concentration

  float c_eq = (h0 - h1);
  if(c_eq < 0.0f){
    c_eq = 0.0f;
  }

  const float vol = vol_b[ind];
  const float sed = sed_b[ind];

  const float effD = depositionRate;

  float c_diff = (c_eq * vol - sed);
  if(isnan(c_diff)){
    c_diff = 0.0f;
  }

  if(effD * c_diff < 0.0f){
    c_diff = 0.0f;
//    // cap the sediment amount to the available sediment if particle loses
//    if(effD * c_diff < -sed){
//      c_diff = -sed / effD;
//    }
  }

  sed_b[ind] += effD * c_diff;
  vol_b[ind] *= (1.0f - evapRate);

  // add to the height-map the negative value of the sediment...
  // note: this should be weighted - distributed

//  if(!oob(pos0, index) && !oob(pos1, index)){
  const int find = index.flatten(pos0);
  atomicAdd(&height[find], -effD * c_diff);
//  }

}

__global__ void clamp(soil::buffer_t<float> height){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= height.elem()) return;

  if(height[ind] > 256) height[ind] = 256;
  if(height[ind] < -256) height[ind] = -256;

}

void gpu_erode(soil::buffer &buffer, soil::buffer& discharge, const soil::index &index, const size_t steps, const size_t maxage){

  std::cout<<"Launched GPU Erode Kernel"<<std::endl;

  if(buffer.host() != soil::host_t::GPU){
    throw soil::error::mismatch_host(soil::host_t::GPU, buffer.host());
  }

  //
  // Type-Cast Buffers
  //

  auto buffer_t = buffer.as<float>();
  auto index_t = index.as<flat_t<2>>();

  auto discharge_t = discharge.as<float>();

  //
  // Particle Buffers
  //

  std::cout<<"Setting Up Particle Buffers..."<<std::endl;

  const size_t n_particles = 1024;

  soil::buffer_t<vec2> pos_buf(n_particles, soil::host_t::GPU);
  soil::buffer_t<vec2> spd_buf(n_particles, soil::host_t::GPU);

  soil::buffer_t<float> vol_buf(n_particles, soil::host_t::GPU);
  soil::buffer_t<float> sed_buf(n_particles, soil::host_t::GPU);

  //
  // Initialize Rand-State Buffer
  //

  std::cout<<"Initializing Random State..."<<std::endl;

  hiprandState* randStates;
  hipMalloc((void**)&randStates, n_particles * sizeof(hiprandState));
  init_randstate<<<block(n_particles, 512), n_particles>>>(randStates, n_particles, 0);

  //
  // Execute Erosion Loop
  //

  std::cout<<"Eroding..."<<std::endl;

  for(size_t step = 0; step < steps; ++step){

    //
    // Spawn Particles
    //

    spawn<<<block(n_particles, 512), n_particles>>>(pos_buf, randStates, index.as<flat_t<2>>());
    fill<<<block(n_particles, 512), n_particles>>>(spd_buf, vec2(0.0f));
    fill<<<block(n_particles, 512), n_particles>>>(vol_buf, 1.0f);
    fill<<<block(n_particles, 512), n_particles>>>(sed_buf, 0.0f);
    fill<<<block(n_particles, 512), n_particles>>>(discharge_t, 0.0f);

    //
    // Erosion Loop
    //  1. Descend Particles (Accelerate, Move)
    //  2. Mass-Transfer
    //  3. Track

    for(size_t age = 0; age < maxage; ++age){

      descend<<<block(n_particles, 512), 512>>>(buffer_t, index_t, pos_buf, spd_buf);
      // _discharge<<<block(n_particles, 512), 512>>>(discharge_t, index_t, pos_buf, sed_buf);
      transfer<<<block(n_particles, 512), 512>>>(buffer_t, index_t, pos_buf, spd_buf, vol_buf, sed_buf);

    }

  }

  // necessary solution to temporarily fix an indexing problem
  // which is introducing unrealistically large values into the
  // height buffer - who knows why.

  clamp<<<block(buffer_t.elem(), 1024), 1024>>>(buffer_t);

  // Loop for Number of Steps per Particle:
  // 1. Accelerate Particles
  // 2. 

  //
  // Note: In principle we can use an age buffer,
  //  or a termination check, to see if the particle
  //  gets re-spawned immediately and we just keep looping...

}

} // end of namespace soil

#endif