#include "hip/hip_runtime.h"
#ifndef SOILLIB_NODE_EROSION_CU
#define SOILLIB_NODE_EROSION_CU
#define HAS_CUDA

#include <soillib/node/erosion.hpp>
#include <soillib/util/error.hpp>
#include <soillib/node/lerp.cu>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <iostream>

namespace soil {

namespace {

template<typename T>
struct sample_t {
  glm::ivec2 pos;
  T value;
  bool oob = true;
};

template<typename T, typename I>
__device__ void gather(const soil::buffer_t<T> &buffer_t, const I index, glm::ivec2 p, sample_t<T> px[5], sample_t<T> py[5]) {
  for (int i = 0; i < 5; ++i) {

    const glm::ivec2 pos_x = p + glm::ivec2(-2 + i, 0);
    if (!index.oob(pos_x)) {
      px[i].oob = false;
      px[i].pos = pos_x;

      const size_t ind = index.flatten(pos_x);
      px[i].value = buffer_t[ind];
    }

    const glm::ivec2 pos_y = p + glm::ivec2(0, -2 + i);
    if (!index.oob(pos_y)) {
      py[i].oob = false;
      py[i].pos = pos_y;

      const size_t ind = index.flatten(pos_y);
      py[i].value = buffer_t[ind];
    }
  }
}

template<std::floating_point T>
__device__ glm::vec2 gradient_detailed(sample_t<T> px[5], sample_t<T> py[5]) {

  glm::vec2 g = glm::vec2(0, 0);

  // X-Element
  if (!px[0].oob && !px[4].oob)
    g.x = (1.0f * px[0].value - 8.0f * px[1].value + 8.0f * px[3].value - 1.0f * px[4].value) / 12.0f;

  else if (!px[0].oob && !px[3].oob)
    g.x = (1.0f * px[0].value - 6.0f * px[1].value + 3.0f * px[2].value + 2.0f * px[3].value) / 6.0f;

  else if (!px[0].oob && !px[2].oob)
    g.x = (1.0f * px[0].value - 4.0f * px[1].value + 3.0f * px[2].value) / 2.0f;

  else if (!px[1].oob && !px[4].oob)
    g.x = (-2.0f * px[1].value - 3.0f * px[2].value + 6.0f * px[3].value - 1.0f * px[4].value) / 6.0f;

  else if (!px[2].oob && !px[4].oob)
    g.x = (-3.0f * px[2].value + 4.0f * px[3].value - 1.0f * px[4].value) / 2.0f;

  else if (!px[1].oob && !px[3].oob)
    g.x = (-1.0f * px[1].value + 1.0f * px[3].value) / 2.0f;

  else if (!px[2].oob && !px[3].oob)
    g.x = (-1.0f * px[2].value + 1.0f * px[3].value) / 1.0f;

  else if (!px[1].oob && !px[2].oob)
    g.x = (-1.0f * px[1].value + 1.0f * px[2].value) / 1.0f;

  // Y-Element

  if (!py[0].oob && !py[4].oob)
    g.y = (1.0f * py[0].value - 8.0f * py[1].value + 8.0f * py[3].value - 1.0f * py[4].value) / 12.0f;

  else if (!py[0].oob && !py[3].oob)
    g.y = (1.0f * py[0].value - 6.0f * py[1].value + 3.0f * py[2].value + 2.0f * py[3].value) / 6.0f;

  else if (!py[0].oob && !py[2].oob)
    g.y = (1.0f * py[0].value - 4.0f * py[1].value + 3.0f * py[2].value) / 2.0f;

  else if (!py[1].oob && !py[4].oob)
    g.y = (-2.0f * py[1].value - 3.0f * py[2].value + 6.0f * py[3].value - 1.0f * py[4].value) / 6.0f;

  else if (!py[2].oob && !py[4].oob)
    g.y = (-3.0f * py[2].value + 4.0f * py[3].value - 1.0f * py[4].value) / 2.0f;

  else if (!py[1].oob && !py[3].oob)
    g.y = (-1.0f * py[1].value + 1.0f * py[3].value) / 2.0f;

  else if (!py[2].oob && !py[3].oob)
    g.y = (-1.0f * py[2].value + 1.0f * py[3].value) / 1.0f;

  else if (!py[1].oob && !py[2].oob)
    g.y = (-1.0f * py[1].value + 1.0f * py[2].value) / 1.0f;

  return g;
}

int block(const int elem, const int thread){
  return (elem + thread - 1)/thread;
}

}

__global__ void init_randstate(hiprandState* states, const size_t N, const size_t seed) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= N) return;
  hiprand_init(seed, index, 0, &states[index]);
}

__global__ void spawn(buffer_t<vec2> pos_buf, hiprandState* randStates, flat_t<2> index){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= pos_buf.size()) return;

  hiprandState* randState = &randStates[n];
  vec2 pos {
    hiprand_uniform(randState)*float(index[0]),
    hiprand_uniform(randState)*float(index[1])
  };

  pos_buf[n] = pos;

}

template<typename T>
__global__ void fill(soil::buffer_t<T> buf, const T val){
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < buf.size())
    buf[index] = val;
}

__global__ void descend(const soil::buffer_t<float> height, const soil::buffer_t<float> discharge_b, const soil::buffer_t<vec2> momentum_b, const soil::flat_t<2> index, soil::buffer_t<vec2> pos, soil::buffer_t<vec2> speed, soil::buffer_t<float> vol_b, soil::buffer_t<float> sed_b){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= pos.elem()) return;

  if(index.oob(pos[ind])){
    return;
  }

  if(oob(pos[ind], index)){
    return;
  }

  sample_t<float> px[5], py[5];
  gather<float, soil::flat_t<2>>(height, index, ivec2(pos[ind]), px, py);
  const vec2 grad = gradient_detailed<float>(px, py);
  const vec3 normal = glm::normalize(vec3(-grad.x, -grad.y, 1.0f));

  // Speed Update

  // Gravity 

  const float gravity = 2.0f;
  const float volume = vol_b[ind];

  const float minVol = 0.001;
  if (volume < minVol) {
    return;
  }

  vec2 s = speed[ind];
  s += gravity * vec2(normal.x, normal.y) / volume;

  // Momentum Transfer

//  const vec2 fspeed = momentum_b[ind];
//  const float discharge = erf(0.4f * discharge_b[ind]);
//  const float momentumTransfer = 2.0f;
//  if (glm::length(fspeed) > 0 && glm::length(s) > 0)
//    s += momentumTransfer * glm::dot(glm::normalize(fspeed), glm::normalize(s)) / (volume + discharge) * fspeed;
  
  // Normalize Time-Step, Increment
  
  if(glm::length(s) > 0.0f){
    s = sqrtf(2.0f) * glm::normalize(s);
  }

  speed[ind] = s;
  pos[ind] += s;

}

__global__ void track(soil::buffer_t<float> discharge, soil::buffer_t<vec2> momentum, const soil::flat_t<2> index, soil::buffer_t<vec2> pos, soil::buffer_t<vec2> speed, soil::buffer_t<float> vol_b){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= pos.elem()) return;

  if(index.oob(pos[ind])) 
    return;

  const int find = index.flatten(pos[ind]);
  const float vol = vol_b[ind];
  atomicAdd(&discharge[find], vol);

  const vec2 m = vol * speed[ind];
  atomicAdd(&momentum[find].x, m.x);
  atomicAdd(&momentum[find].y, m.y);
}

__global__ void transfer(soil::buffer_t<float> height, const soil::buffer_t<float> discharge_b, const soil::flat_t<2> index, soil::buffer_t<vec2> pos_b, soil::buffer_t<vec2> speed_b, soil::buffer_t<float> vol_b, soil::buffer_t<float> sed_b){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= pos_b.elem()) return;

  const float evapRate = 0.001f;
  const float depositionRate = 0.05f;

  const vec2 speed = speed_b[ind];  // Current Speed
  const vec2 pos1 = pos_b[ind];     // Current Position
  const vec2 pos0 = pos1 - speed;   // Old Position

  if(index.oob(pos0)) return;
  if(index.flatten(pos0) == 0) return;

  // Sample Height Values (Old Position, New Position)
  
  float h0 = height[index.flatten(pos0)];
  float h1 = 0.99f*h0;
  if(!index.oob(pos1)){
    h1 = height[index.flatten(pos1)];
  }

  if(isnan(h0) || isnan(h1)){
    return;
  }

  // Compute Equilibrium Mass-Transfer

  const float vol = vol_b[ind]; // Water Volume
  const float sed = sed_b[ind]; // Sediment Mass

  // Equilibrium Concentration
  // Note: Can't be Negative!
  const float entrainment = 10.0f;
  float discharge = erf(0.4f * discharge_b[index.flatten(pos0)]);
  // if(isnan(discharge))
  discharge = 0.0f;
  
  const float c_eq = glm::max(h0 - h1, 0.0f) * (1.0f + discharge * entrainment);
  const float effD = depositionRate;

  float c_diff = (c_eq * vol - sed);
  if(isnan(c_diff)){
    c_diff = 0.0f;
  }

  // can only give as much mass as we have...
  if(effD * c_diff < -sed){
    c_diff = -sed / effD;
  }

  // Execute Mass-Transfer
  const int find = index.flatten(ivec2(pos0));

  //!\todo figure out why find zero gives so many problems...
  // why would this every be a problem? I don't get it...
  if(find != 0){

    sed_b[ind] += effD * c_diff;
    vol_b[ind] *= (1.0f - evapRate);

    atomicAdd(&height[find], -effD * c_diff);

  }
}

template<typename T>
__global__ void filter(soil::buffer_t<T>& discharge, const soil::buffer_t<T>& discharge_track, float lrate){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= discharge.elem()) return;

  const T val = discharge[ind];
  discharge[ind] = val * (1.0f - lrate) + discharge_track[ind] * lrate;

}

__global__ void clamp(soil::buffer_t<float> height){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= height.elem()) return;

  if(height[ind] > 256) height[ind] = 256;
  if(height[ind] < -256) height[ind] = -256;

}

void gpu_erode(soil::buffer &buffer, soil::buffer& discharge, soil::buffer& momentum, const soil::index &index, const size_t steps, const size_t maxage){

  std::cout<<"Launched GPU Erode Kernel"<<std::endl;

  if(buffer.host() != soil::host_t::GPU){
    throw soil::error::mismatch_host(soil::host_t::GPU, buffer.host());
  }

  //
  // Type-Cast Buffers
  //

  auto buffer_t = buffer.as<float>();
  auto index_t = index.as<flat_t<2>>();

  auto discharge_t = discharge.as<float>();
  auto momentum_t = momentum.as<vec2>();

  //
  // Particle Buffers
  //

  std::cout<<"Setting Up Particle Buffers..."<<std::endl;

  const size_t n_particles = 1024;

  soil::buffer_t<vec2> pos_buf(n_particles, soil::host_t::GPU);
  soil::buffer_t<vec2> spd_buf(n_particles, soil::host_t::GPU);

  soil::buffer_t<float> vol_buf(n_particles, soil::host_t::GPU);
  soil::buffer_t<float> sed_buf(n_particles, soil::host_t::GPU);

  soil::buffer_t<float> discharge_track(discharge.elem(), soil::host_t::GPU);
  soil::buffer_t<vec2> momentum_track(momentum.elem(), soil::host_t::GPU);

  //
  // Initialize Rand-State Buffer
  //

  std::cout<<"Initializing Random State..."<<std::endl;

  hiprandState* randStates;
  hipMalloc((void**)&randStates, n_particles * sizeof(hiprandState));
  init_randstate<<<block(n_particles, 512), n_particles>>>(randStates, n_particles, 0);

  //
  // Execute Erosion Loop
  //

  std::cout<<"Eroding..."<<std::endl;

  for(size_t step = 0; step < steps; ++step){

    //
    // Spawn Particles
    //

    spawn<<<block(n_particles, 512), n_particles>>>(pos_buf, randStates, index.as<flat_t<2>>());
    fill<<<block(n_particles, 512), n_particles>>>(spd_buf, vec2(0.0f));
    fill<<<block(n_particles, 512), n_particles>>>(vol_buf, 1.0f);
    fill<<<block(n_particles, 512), n_particles>>>(sed_buf, 0.0f);

    fill<<<block(discharge_track.elem(), 1024), 1024>>>(discharge_track, 0.0f);
    fill<<<block(momentum_track.elem(), 1024), 1024>>>(momentum_track, vec2(0.0f));

    //
    // Erosion Loop
    //  1. Descend Particles (Accelerate, Move)
    //  2. Mass-Transfer
    //  3. Track

    for(size_t age = 0; age < maxage; ++age){

      descend<<<block(n_particles, 512), 512>>>(buffer_t, discharge_track, momentum_track, index_t, pos_buf, spd_buf, vol_buf, sed_buf);
      hipDeviceSynchronize();
//
//      track<<<block(n_particles, 512), 512>>>(discharge_track, momentum_track, index_t, pos_buf, spd_buf, vol_buf);
//      hipDeviceSynchronize();

      transfer<<<block(n_particles, 512), 512>>>(buffer_t, discharge_track, index_t, pos_buf, spd_buf, vol_buf, sed_buf);
      hipDeviceSynchronize();

    }

//    filter<<<block(index.elem(), 1024), 1024>>>(discharge_t, discharge_track, 0.01f);
//    filter<<<block(index.elem(), 1024), 1024>>>(momentum_t, momentum_track, 0.01f);
//    hipDeviceSynchronize();
  
  }

  // necessary solution to temporarily fix an indexing problem
  // which is introducing unrealistically large values into the
  // height buffer - who knows why.

  clamp<<<block(buffer_t.elem(), 1024), 1024>>>(buffer_t);

  // Loop for Number of Steps per Particle:
  // 1. Accelerate Particles
  // 2. 

  //
  // Note: In principle we can use an age buffer,
  //  or a termination check, to see if the particle
  //  gets re-spawned immediately and we just keep looping...

}

} // end of namespace soil

#endif