#include "hip/hip_runtime.h"
#ifndef SOILLIB_NODE_EROSION_CU
#define SOILLIB_NODE_EROSION_CU
#define HAS_CUDA

#include <soillib/node/erosion.hpp>
#include <soillib/util/error.hpp>
#include <soillib/node/lerp.cu>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <iostream>

namespace soil {

namespace {

template<typename T>
struct sample_t {
  glm::ivec2 pos;
  T value;
  bool oob = true;
};

template<typename T, typename I>
__device__ void gather(const soil::buffer_t<T> &buffer_t, const I index, glm::ivec2 p, sample_t<T> px[5], sample_t<T> py[5]) {
  for (int i = 0; i < 5; ++i) {

    const glm::ivec2 pos_x = p + glm::ivec2(-2 + i, 0);
    if (!index.oob(pos_x)) {
      px[i].oob = false;
      px[i].pos = pos_x;

      const size_t ind = index.flatten(pos_x);
      px[i].value = buffer_t[ind];
    }

    const glm::ivec2 pos_y = p + glm::ivec2(0, -2 + i);
    if (!index.oob(pos_y)) {
      py[i].oob = false;
      py[i].pos = pos_y;

      const size_t ind = index.flatten(pos_y);
      py[i].value = buffer_t[ind];
    }
  }
}

template<std::floating_point T>
__device__ glm::vec2 gradient_detailed(sample_t<T> px[5], sample_t<T> py[5]) {

  glm::vec2 g = glm::vec2(0, 0);

  // X-Element
  if (!px[0].oob && !px[4].oob)
    g.x = (1.0f * px[0].value - 8.0f * px[1].value + 8.0f * px[3].value - 1.0f * px[4].value) / 12.0f;

  else if (!px[0].oob && !px[3].oob)
    g.x = (1.0f * px[0].value - 6.0f * px[1].value + 3.0f * px[2].value + 2.0f * px[3].value) / 6.0f;

  else if (!px[0].oob && !px[2].oob)
    g.x = (1.0f * px[0].value - 4.0f * px[1].value + 3.0f * px[2].value) / 2.0f;

  else if (!px[1].oob && !px[4].oob)
    g.x = (-2.0f * px[1].value - 3.0f * px[2].value + 6.0f * px[3].value - 1.0f * px[4].value) / 6.0f;

  else if (!px[2].oob && !px[4].oob)
    g.x = (-3.0f * px[2].value + 4.0f * px[3].value - 1.0f * px[4].value) / 2.0f;

  else if (!px[1].oob && !px[3].oob)
    g.x = (-1.0f * px[1].value + 1.0f * px[3].value) / 2.0f;

  else if (!px[2].oob && !px[3].oob)
    g.x = (-1.0f * px[2].value + 1.0f * px[3].value) / 1.0f;

  else if (!px[1].oob && !px[2].oob)
    g.x = (-1.0f * px[1].value + 1.0f * px[2].value) / 1.0f;

  // Y-Element

  if (!py[0].oob && !py[4].oob)
    g.y = (1.0f * py[0].value - 8.0f * py[1].value + 8.0f * py[3].value - 1.0f * py[4].value) / 12.0f;

  else if (!py[0].oob && !py[3].oob)
    g.y = (1.0f * py[0].value - 6.0f * py[1].value + 3.0f * py[2].value + 2.0f * py[3].value) / 6.0f;

  else if (!py[0].oob && !py[2].oob)
    g.y = (1.0f * py[0].value - 4.0f * py[1].value + 3.0f * py[2].value) / 2.0f;

  else if (!py[1].oob && !py[4].oob)
    g.y = (-2.0f * py[1].value - 3.0f * py[2].value + 6.0f * py[3].value - 1.0f * py[4].value) / 6.0f;

  else if (!py[2].oob && !py[4].oob)
    g.y = (-3.0f * py[2].value + 4.0f * py[3].value - 1.0f * py[4].value) / 2.0f;

  else if (!py[1].oob && !py[3].oob)
    g.y = (-1.0f * py[1].value + 1.0f * py[3].value) / 2.0f;

  else if (!py[2].oob && !py[3].oob)
    g.y = (-1.0f * py[2].value + 1.0f * py[3].value) / 1.0f;

  else if (!py[1].oob && !py[2].oob)
    g.y = (-1.0f * py[1].value + 1.0f * py[2].value) / 1.0f;

  return g;
}

int block(const int elem, const int thread){
  return (elem + thread - 1)/thread;
}

}

__global__ void init_randstate(hiprandState* states, const size_t N, const size_t seed) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= N) return;
  hiprand_init(seed, index, 0, &states[index]);
}

__global__ void spawn(buffer_t<vec2> pos_buf, hiprandState* randStates, flat_t<2> index){

  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= pos_buf.size()) return;

  hiprandState* randState = &randStates[n];
  vec2 pos {
    hiprand_uniform(randState)*float(index[0]),
    hiprand_uniform(randState)*float(index[1])
  };

  pos_buf[n] = pos;

}

template<typename T>
__global__ void fill(soil::buffer_t<T> buf, const T val){
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < buf.size())
    buf[index] = val;
}

__global__ void descend(const soil::buffer_t<float> height, const soil::flat_t<2> index, soil::buffer_t<vec2> pos, soil::buffer_t<vec2> speed, soil::buffer_t<float> vol_b, soil::buffer_t<float> sed_b){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= pos.elem()) return;

  if(index.oob(pos[ind])){
    return;
  }

  if(oob(pos[ind], index)){
    return;
  }

  sample_t<float> px[5], py[5];
  gather<float, soil::flat_t<2>>(height, index, ivec2(pos[ind]), px, py);
  const vec2 grad = gradient_detailed<float>(px, py);

  // Speed Update

  vec2 s = speed[ind];
  s += sqrtf(2.0f) * glm::normalize(-vec2(grad.x, grad.y));
  if(glm::length(s) > 0.0f){
    s = sqrtf(2.0f) * glm::normalize(s);
  }

  speed[ind] = s;
  pos[ind] += s;

}

__global__ void _discharge(soil::buffer_t<float> discharge, const soil::flat_t<2> index, soil::buffer_t<vec2> pos, soil::buffer_t<float> vol_b){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= pos.elem()) return;

  if(!index.oob(pos[ind])) 
    return;

  const int find = index.flatten(pos[ind]);
  const float vol = vol_b[ind];
  atomicAdd(&discharge[find], vol);

}

__global__ void transfer(soil::buffer_t<float> height, const soil::flat_t<2> index, soil::buffer_t<vec2> pos_b, soil::buffer_t<vec2> speed_b, soil::buffer_t<float> vol_b, soil::buffer_t<float> sed_b){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= pos_b.elem()) return;

  const float evapRate = 0.001f;
  const float depositionRate = 0.05f;

  const vec2 speed = speed_b[ind];  // Current Speed
  const vec2 pos1 = pos_b[ind];     // Current Position
  const vec2 pos0 = pos1 - speed;   // Old Position

  if(index.oob(pos0)) return;

  // Sample Height Values (Old Position, New Position)
  
  float h0 = height[index.flatten(pos0)];
  float h1 = 0.99f*h0;
  if(!index.oob(pos1)){
    h1 = height[index.flatten(pos1)];
  }

  if(isnan(h0) || isnan(h1)){
    return;
  }

  // Compute Equilibrium Mass-Transfer

  const float vol = vol_b[ind]; // Water Volume
  const float sed = sed_b[ind]; // Sediment Mass

  // Equilibrium Concentration
  // Note: Can't be Negative!
  const float c_eq = glm::max(h0 - h1, 0.0f);
  const float effD = depositionRate;

  float c_diff = (c_eq * vol - sed);
  if(isnan(c_diff)){
    c_diff = 0.0f;
  }

  // can only give as much mass as we have...
  if(effD * c_diff < -sed){
    c_diff = -sed / effD;
  }

  // Execute Mass-Transfer
  const int find = index.flatten(ivec2(pos0));

  //!\todo figure out why find zero gives so many problems...
  // why would this every be a problem? I don't get it...
  if(find != 0){

    sed_b[ind] += effD * c_diff;
    vol_b[ind] *= (1.0f - evapRate);

    atomicAdd(&height[find], -effD * c_diff);

  }
}

__global__ void clamp(soil::buffer_t<float> height){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= height.elem()) return;

  if(height[ind] > 256) height[ind] = 256;
  if(height[ind] < -256) height[ind] = -256;

}

void gpu_erode(soil::buffer &buffer, soil::buffer& discharge, const soil::index &index, const size_t steps, const size_t maxage){

  std::cout<<"Launched GPU Erode Kernel"<<std::endl;

  if(buffer.host() != soil::host_t::GPU){
    throw soil::error::mismatch_host(soil::host_t::GPU, buffer.host());
  }

  //
  // Type-Cast Buffers
  //

  auto buffer_t = buffer.as<float>();
  auto index_t = index.as<flat_t<2>>();

  auto discharge_t = discharge.as<float>();

  //
  // Particle Buffers
  //

  std::cout<<"Setting Up Particle Buffers..."<<std::endl;

  const size_t n_particles = 1024;

  soil::buffer_t<vec2> pos_buf(n_particles, soil::host_t::GPU);
  soil::buffer_t<vec2> spd_buf(n_particles, soil::host_t::GPU);

  soil::buffer_t<float> vol_buf(n_particles, soil::host_t::GPU);
  soil::buffer_t<float> sed_buf(n_particles, soil::host_t::GPU);

  //
  // Initialize Rand-State Buffer
  //

  std::cout<<"Initializing Random State..."<<std::endl;

  hiprandState* randStates;
  hipMalloc((void**)&randStates, n_particles * sizeof(hiprandState));
  init_randstate<<<block(n_particles, 512), n_particles>>>(randStates, n_particles, 0);

  //
  // Execute Erosion Loop
  //

  std::cout<<"Eroding..."<<std::endl;

  for(size_t step = 0; step < steps; ++step){

    //
    // Spawn Particles
    //

    spawn<<<block(n_particles, 512), n_particles>>>(pos_buf, randStates, index.as<flat_t<2>>());
    fill<<<block(n_particles, 512), n_particles>>>(spd_buf, vec2(0.0f));
    fill<<<block(n_particles, 512), n_particles>>>(vol_buf, 1.0f);
    fill<<<block(n_particles, 512), n_particles>>>(sed_buf, 0.0f);
    fill<<<block(n_particles, 512), n_particles>>>(discharge_t, 0.0f);

    //
    // Erosion Loop
    //  1. Descend Particles (Accelerate, Move)
    //  2. Mass-Transfer
    //  3. Track

    for(size_t age = 0; age < maxage; ++age){

      descend<<<block(n_particles, 512), 512>>>(buffer_t, index_t, pos_buf, spd_buf, vol_buf, sed_buf);
//      _discharge<<<block(n_particles, 512), 512>>>(discharge_t, index_t, pos_buf, vol_buf);
      transfer<<<block(n_particles, 512), 512>>>(buffer_t, index_t, pos_buf, spd_buf, vol_buf, sed_buf);

    }

  }

  // necessary solution to temporarily fix an indexing problem
  // which is introducing unrealistically large values into the
  // height buffer - who knows why.

  clamp<<<block(buffer_t.elem(), 1024), 1024>>>(buffer_t);

  // Loop for Number of Steps per Particle:
  // 1. Accelerate Particles
  // 2. 

  //
  // Note: In principle we can use an age buffer,
  //  or a termination check, to see if the particle
  //  gets re-spawned immediately and we just keep looping...

}

} // end of namespace soil

#endif