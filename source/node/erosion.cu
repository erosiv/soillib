#include "hip/hip_runtime.h"
#ifndef SOILLIB_NODE_EROSION_CU
#define SOILLIB_NODE_EROSION_CU
#define HAS_CUDA

#include <soillib/node/erosion.hpp>
#include <soillib/util/error.hpp>
#include <soillib/node/lerp.cu>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <iostream>

namespace soil {

namespace {

template<typename T>
struct sample_t {
  glm::ivec2 pos;
  T value;
  bool oob = true;
};

template<typename T, typename I>
__device__ void gather(const soil::buffer_t<T> &buffer_t, const I index, glm::ivec2 p, sample_t<T> px[5], sample_t<T> py[5]) {
  for (int i = 0; i < 5; ++i) {

    const glm::ivec2 pos_x = p + glm::ivec2(-2 + i, 0);
    if (!index.oob(pos_x)) {
      px[i].oob = false;
      px[i].pos = pos_x;

      const size_t ind = index.flatten(pos_x);
      px[i].value = buffer_t[ind];
    }

    const glm::ivec2 pos_y = p + glm::ivec2(0, -2 + i);
    if (!index.oob(pos_y)) {
      py[i].oob = false;
      py[i].pos = pos_y;

      const size_t ind = index.flatten(pos_y);
      py[i].value = buffer_t[ind];
    }
  }
}

template<std::floating_point T>
__device__ glm::vec2 gradient_detailed(sample_t<T> px[5], sample_t<T> py[5]) {

  glm::vec2 g = glm::vec2(0, 0);

  // X-Element
  if (!px[0].oob && !px[4].oob)
    g.x = (1.0f * px[0].value - 8.0f * px[1].value + 8.0f * px[3].value - 1.0f * px[4].value) / 12.0f;

  else if (!px[0].oob && !px[3].oob)
    g.x = (1.0f * px[0].value - 6.0f * px[1].value + 3.0f * px[2].value + 2.0f * px[3].value) / 6.0f;

  else if (!px[0].oob && !px[2].oob)
    g.x = (1.0f * px[0].value - 4.0f * px[1].value + 3.0f * px[2].value) / 2.0f;

  else if (!px[1].oob && !px[4].oob)
    g.x = (-2.0f * px[1].value - 3.0f * px[2].value + 6.0f * px[3].value - 1.0f * px[4].value) / 6.0f;

  else if (!px[2].oob && !px[4].oob)
    g.x = (-3.0f * px[2].value + 4.0f * px[3].value - 1.0f * px[4].value) / 2.0f;

  else if (!px[1].oob && !px[3].oob)
    g.x = (-1.0f * px[1].value + 1.0f * px[3].value) / 2.0f;

  else if (!px[2].oob && !px[3].oob)
    g.x = (-1.0f * px[2].value + 1.0f * px[3].value) / 1.0f;

  else if (!px[1].oob && !px[2].oob)
    g.x = (-1.0f * px[1].value + 1.0f * px[2].value) / 1.0f;

  // Y-Element

  if (!py[0].oob && !py[4].oob)
    g.y = (1.0f * py[0].value - 8.0f * py[1].value + 8.0f * py[3].value - 1.0f * py[4].value) / 12.0f;

  else if (!py[0].oob && !py[3].oob)
    g.y = (1.0f * py[0].value - 6.0f * py[1].value + 3.0f * py[2].value + 2.0f * py[3].value) / 6.0f;

  else if (!py[0].oob && !py[2].oob)
    g.y = (1.0f * py[0].value - 4.0f * py[1].value + 3.0f * py[2].value) / 2.0f;

  else if (!py[1].oob && !py[4].oob)
    g.y = (-2.0f * py[1].value - 3.0f * py[2].value + 6.0f * py[3].value - 1.0f * py[4].value) / 6.0f;

  else if (!py[2].oob && !py[4].oob)
    g.y = (-3.0f * py[2].value + 4.0f * py[3].value - 1.0f * py[4].value) / 2.0f;

  else if (!py[1].oob && !py[3].oob)
    g.y = (-1.0f * py[1].value + 1.0f * py[3].value) / 2.0f;

  else if (!py[2].oob && !py[3].oob)
    g.y = (-1.0f * py[2].value + 1.0f * py[3].value) / 1.0f;

  else if (!py[1].oob && !py[2].oob)
    g.y = (-1.0f * py[1].value + 1.0f * py[2].value) / 1.0f;

  return g;
}

int block(const int elem, const int thread){
  return (elem + thread - 1)/thread;
}

}

//
// Utility Kernels
//

template<typename T>
__global__ void fill(soil::buffer_t<T> buf, const T val){
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= buf.elem()) return;
  buf[index] = val;
}

__global__ void init_randstate(hiprandState* states, const size_t N, const size_t seed) {
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= N) return;
  hiprand_init(seed, n, 0, &states[n]);
}

__global__ void spawn(buffer_t<vec2> pos, hiprandState* randStates, flat_t<2> index){
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= pos.elem()) return;

  hiprandState* randState = &randStates[n];
  pos[n] = vec2{
    hiprand_uniform(randState)*float(index[0]),
    hiprand_uniform(randState)*float(index[1])
  };
}

__global__ void filter(soil::buffer_t<float> buffer, const soil::buffer_t<float> buffer_track, const float lrate){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= buffer.elem()) return;
  if(ind >= buffer_track.elem()) return;

  float val = buffer[ind];
  float val_track = buffer_track[ind];
  buffer[ind] = val * (1.0f - lrate) +  val_track * lrate;
}

__global__ void filter(soil::buffer_t<vec2> buffer, const soil::buffer_t<vec2> buffer_track, const float lrate){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= buffer.elem()) return;
  if(ind >= buffer_track.elem()) return;

  vec2 val = buffer[ind];
  vec2 val_track = buffer_track[ind];
  buffer[ind] = val * (1.0f - lrate) +  val_track * lrate;
}

__global__ void track(model_t model, soil::buffer_t<float> discharge_track, soil::buffer_t<vec2> momentum_track, particle_t particles){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= particles.elem) return;

  const ivec2 pos = particles.pos[ind];
  if(model.index.oob(pos)) return;

  // why would I have to scale this by lrate?

  const int find = model.index.flatten(pos);
  const float vol = particles.vol[ind];
  atomicAdd(&discharge_track[find], vol);   // Accumulate Current Volume into Tracking Buffer

  const vec2 m = vol * particles.spd[ind];
  atomicAdd(&momentum_track[find].x, m.x);
  atomicAdd(&momentum_track[find].y, m.y);

}

__global__ void _normalize(soil::buffer_t<float> out, const float P){
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= out.elem()) return;
  out[n] = 1.0f + P * out[n];
}

//
// Cascading Kernel
//
// Effectively we have to compute the height-difference between every cell
// and its non-out-of-bounds neighbors, then we have to transfer the sediment.
// How do we do this without race conditions?
// I suppose that we need an additional buffer to determine the updated sediment amounts...
// so that we can ping-pong back and forth...

// for now, we will implement this as a device function locally and perhaps switch to
// a singular kernel later.

// __global__ void cascade(model_t model){
// }

__device__ void cascade(model_t& model, const glm::ivec2 ipos, buffer_t<float>& transfer, const param_t param) {

  if(model.index.oob(ipos))
    return;

  // Get Non-Out-of-Bounds Neighbors

  const glm::ivec2 n[] = {
    glm::ivec2(-1, -1),
    glm::ivec2(-1, 0),
    glm::ivec2(-1, 1),
    glm::ivec2(0, -1),
    glm::ivec2(0, 1),
    glm::ivec2(1, -1),
    glm::ivec2(1, 0),
    glm::ivec2(1, 1)
  };

  struct Point {
    glm::ivec2 pos;
    float h;
    float d;
  } sn[8];

  int num = 0;

  for(auto &nn : n){

    glm::ivec2 npos = ipos + nn;

    if(model.index.oob(npos))
      continue;

    const size_t index = model.index.flatten(npos);
    const float height = model.height[index];
    sn[num] = {npos, height, length(glm::vec2(nn))};
    ++num;
  }

  const size_t index = model.index.flatten(ipos);
  const float height = model.height[index];
  float h_ave = height;
  // for (int i = 0; i < num; ++i)
  //   h_ave += sn[i].h;
  // h_ave /= (float)(num + 1);

  float transfer_tot = 0.0f;
  
  for(int i = 0; i < num; ++i){

    // Full Height-Different Between Positions!
    float diff = h_ave - sn[i].h;
    if (diff == 0) // No Height Difference
      continue;

    // The Amount of Excess Difference!
    float excess = 0.0f;
    excess = abs(diff) - sn[i].d * param.maxdiff;
    if (excess <= 0) // No Excess
      continue;

    excess = (diff > 0) ? -excess : excess;

    // Actual Amount Transferred
    float transfer = param.settling * excess / 2.0f;
    transfer_tot += transfer;
  }

  transfer[index] = transfer_tot / (float) num;
}

__global__ void compute_cascade(model_t model, buffer_t<float> transfer, const param_t param){

  // note: this should be two kernels.
  // one to compute the amount of sediment
  // that has to be transferred -
  // one to then actual transfer it.

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= model.elem) return;
  const ivec2 ipos = model.index.unflatten(ind);
  cascade(model, ipos, transfer, param);

}

__global__ void apply_cascade(model_t model, buffer_t<float> transfer_b, const param_t param){
  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= model.elem) return;

  // Only cascade where agitation exists?

  const float transfer = transfer_b[ind];
  //const float discharge = erf(0.4f * model.discharge[ind]);
  //model.height[ind] += discharge * transfer;
  model.height[ind] += transfer;
}

//
// Erosion Kernels
//

__global__ void descend(const model_t model, particle_t particles, const param_t param){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= particles.elem) return;

  // Retrieve Position, Check Bounds

  const vec2 pos = particles.pos[ind];
  if(model.index.oob(pos))
    return;

  const int find = model.index.flatten(pos);

  // Skip Depleted Particles

  const float volume = particles.vol[ind];
  if (volume < param.minVol) {
    return;
  }

  // Compute Speed Update

  vec2 speed = particles.spd[ind];

  // Gravity Contribution
  // Compute Normal Vector

  sample_t<float> px[5], py[5];
  gather<float, soil::flat_t<2>>(model.height, model.index, ivec2(pos), px, py);
  const vec2 grad = gradient_detailed<float>(px, py);
  const vec3 normal = glm::normalize(vec3(-grad.x, -grad.y, 1.0f));
  speed += param.gravity * vec2(normal.x, normal.y) / volume;

  // Momentum Transfer
  // Volume-Weighted Average of Momentum

  const vec2 fspeed = model.momentum[find];
  const float discharge = model.discharge[find];
  speed += param.momentumTransfer / (volume + discharge) * (fspeed + speed * volume);

  // Normalize Time-Step, Increment
  
  if(glm::length(speed) > 0.0){
    speed = sqrtf(2.0f) * glm::normalize(speed);
  }

  // Compute Slope

  particles.spd[ind] = speed;

  float h0 = model.height[find];
  // note: taking fraction doesn't work for negative numbers...
  // also means that steeper becomes steeper, less-steep less steep.
  // so in general has a not-nice divergent behavior.
  float h1 = h0 - param.exitSlope; 
  if(!model.index.oob(pos + speed)){
    h1 = model.height[model.index.flatten(pos + speed)];
  }
  particles.slope[ind] = (h0 - h1);
}

__global__ void transfer(model_t model, particle_t particles, const param_t param){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= particles.elem) return;

  const vec2 pos = particles.pos[ind];    // Current Position
  if(model.index.oob(pos))
    return;
  
  const int find = model.index.flatten(pos);

  // Compute Equilibrium Mass-Transfer

  const vec2 speed = particles.spd[ind];    // Current Speed
  const float hdiff = particles.slope[ind]; // Local Slope
  const float vol = particles.vol[ind];     // Water Volume
  const float sed = particles.sed[ind];     // Sediment Mass

  // Equilibrium Concentration
  // Note: Can't be Negative!
  const float discharge = log(1.0f + model.discharge[find]);
  const float c_eq = glm::max(hdiff, 0.0f) * (1.0f + discharge * param.entrainment);
  const float effD = param.depositionRate;

  float c_diff = (c_eq * vol - sed);
  if(effD * c_diff < -sed){
    c_diff = -sed / effD;
  }

  // Execute Mass-Transfer

  particles.sed[ind] += effD * c_diff;
  particles.vol[ind] *= (1.0f - param.evapRate);
  atomicAdd(&model.height[find], -effD * c_diff);

  particles.pos[ind] += speed;
}

/*
__global__ void dump(model_t model, particle_t particles, const param_t param){

  const unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if(ind >= particles.elem) return;

  const vec2 pos = particles.pos[ind];    // Current Position
  if(model.index.oob(pos))
    return;

  const int find = model.index.flatten(pos);
  const float sed = particles.sed[ind];     // Sediment Mass

  atomicAdd(&model.height[find], sed);
  particles.sed[ind] = 0.0f;
}
*/

//
// Erosion Function
//

void gpu_erode(model_t& model, const param_t param, const size_t steps, const size_t n_samples){

  if(model.height.host() != soil::host_t::GPU){
    throw soil::error::mismatch_host(soil::host_t::GPU, model.height.host());
  }

  if(model.discharge.host() != soil::host_t::GPU){
    throw soil::error::mismatch_host(soil::host_t::GPU, model.discharge.host());
  }

  if(model.momentum.host() != soil::host_t::GPU){
    throw soil::error::mismatch_host(soil::host_t::GPU, model.momentum.host());
  }

  //
  // Particle Buffers
  //

  soil::buffer_t<float> discharge_track(model.discharge.elem(), soil::host_t::GPU);
  soil::buffer_t<vec2> momentum_track(model.momentum.elem(), soil::host_t::GPU);

  //! \todo remove this allocation, as well as the randstate allocation
  particle_t particles{n_samples};

  //
  // Initialize Rand-State Buffer
  //

  hiprandState* randStates;
  hipMalloc((void**)&randStates, n_samples * sizeof(hiprandState));
  init_randstate<<<block(n_samples, 512), 512>>>(randStates, n_samples, 0);

  hipDeviceSynchronize();

  //
  // Execute Erosion Loop
  //

  for(size_t step = 0; step < steps; ++step){

    //
    // Spawn Particles
    //

    spawn<<<block(n_samples, 512), 512>>>(particles.pos, randStates, model.index);
    fill<<<block(n_samples, 512), 512>>>(particles.spd, vec2(0.0f));
    fill<<<block(n_samples, 512), 512>>>(particles.vol, 1.0f);
    fill<<<block(n_samples, 512), 512>>>(particles.sed, 0.0f);
    fill<<<block(n_samples, 512), 512>>>(particles.slope, 0.0f);

    fill<<<block(discharge_track.elem(), 1024), 1024>>>(discharge_track, 0.0f);
    fill<<<block(momentum_track.elem(), 1024), 1024>>>(momentum_track, vec2(0.0f));
    hipDeviceSynchronize();

    //
    // Erosion Loop
    //  1. Descend Particles (Accelerate, Move)
    //  2. Mass-Transfer
    //  3. Track

    for(size_t age = 0; age < param.maxage; ++age){

      descend<<<block(n_samples, 512), 512>>>(model, particles, param);
      transfer<<<block(n_samples, 512), 512>>>(model, particles, param);
      track<<<block(n_samples, 512), 512>>>(model, discharge_track, momentum_track, particles);

    }

//    // We have to add the excess sediment...
    // dump<<<block(n_particles, 512), 512>>>(model, particles, param);

    //
    // Normalization and Filtering
    //

    // Normalize the Discharge by Sample Probability
    const float P = float(model.elem)/float(n_samples);
    _normalize<<<block(model.elem, 1024), 1024>>>(discharge_track, P);

    // Filter the Result

    filter<<<block(model.elem, 1024), 1024>>>(model.discharge, discharge_track, param.lrate);
    filter<<<block(model.elem, 1024), 1024>>>(model.momentum, momentum_track, param.lrate);
    hipDeviceSynchronize();

//    std::cout<<model.index[0]<<" "<<model.index[1]<<std::endl;

    // atomic add operations might still be coming in -
    // we have to be done before cascading or this fails...
    // we can't be computing the differences before they are determined...

    compute_cascade<<<block(model.elem, 1024), 1024>>>(model, discharge_track, param);
    apply_cascade<<<block(model.elem, 1024), 1024>>>(model, discharge_track, param);
    hipDeviceSynchronize();

  }

}

} // end of namespace soil

#endif